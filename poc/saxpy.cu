#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) y[idx] = a*x[idx] + y[idx];
}

int main(void) {
    int n = 1 << 20;
    float *x, *y, *d_x, *d_y;

    /* allocate host memory */
    x = (float*)malloc(n*sizeof(float));
    y = (float*)malloc(n*sizeof(float));
    hipMalloc(&d_x, n*sizeof(float));
    hipMalloc(&d_y, n*sizeof(float));

    /* init x,y values */
    for (int i = 0; i < n; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /* copy x, y to GPU device */
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

    /* run saxpy GPU kernel */
    saxpy<<<(n+255)/256, 256>>>(n, 2.0f, d_x, d_y);

    /* copy results back to host */
    hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

    /* print error margin */
    float max_err = 0.0f;
    for (int i = 0; i < n; i++) {
        max_err = max(max_err, abs(y[i] - 4.0f));
    }
    printf("Max error: %f\n", max_err);

    /* free memory */
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}
