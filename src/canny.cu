#include "hip/hip_runtime.h"

#include "canny.h"
#define _USE_MATH_DEFINES
#include <math.h>
#define EDGE 0xFFFF

__global__
void cu_apply_gaussian_filter(pixel_t *in_pixels, pixel_t *out_pixels, int rows, int cols, double *in_kernel) 
{
    //copy kernel array from global memory to a shared array
    __shared__ double kernel[KERNEL_SIZE][KERNEL_SIZE];
    for (int i = 0; i < KERNEL_SIZE; ++i) {
        for (int j = 0; j < KERNEL_SIZE; ++j) {
            kernel[i][j] = in_kernel[i*KERNEL_SIZE + j];
        }
    }
    
    __syncthreads();

    //determine id of thread which corresponds to an individual pixel
    int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixNum >= 0 && pixNum < rows * cols) {
   
        double kernelSum;
        double redPixelVal;
        double greenPixelVal;
        double bluePixelVal;

        //Apply Kernel to each pixel of image
        for (int i = 0; i < KERNEL_SIZE; ++i) {
            for (int j = 0; j < KERNEL_SIZE; ++j) {    
            
                //check edge cases, if within bounds, apply filter
                if (((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) >= 0)
                    && ((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) <= rows*cols-1)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) >= 0)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) <= (cols-1))) {

                    redPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].red;
                    greenPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].green;
                    bluePixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].blue;
                    kernelSum += kernel[i][j];
                }
            }
        }
        
        //update output image
        out_pixels[pixNum].red = redPixelVal / kernelSum;
        out_pixels[pixNum].green = greenPixelVal / kernelSum;
        out_pixels[pixNum].blue = bluePixelVal / kernelSum;
    }
}

__device__
void trace_immed_neighbors(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                            unsigned idx, pixel_channel_t t_low, unsigned img_width)
{
    /* directions representing indices of neighbors */
    unsigned n, s, e, w;
    unsigned nw, ne, sw, se;

    /* get indices */
    n = idx - img_width;
    nw = n - 1;
    ne = n + 1;
    s = idx + img_width;
    sw = s - 1;
    se = s + 1;
    w = idx - 1;
    e = idx + 1;

    if ((in_pixels[nw] >= t_low) && (out_pixels[nw] != EDGE)) {
        out_pixels[nw] = EDGE;
    }
    if ((in_pixels[n] >= t_low) && (out_pixels[n] != EDGE)) {
        out_pixels[n] = EDGE;
    }
    if ((in_pixels[ne] >= t_low) && (out_pixels[ne] != EDGE)) {
        out_pixels[ne] = EDGE;
    }
    if ((in_pixels[w] >= t_low) && (out_pixels[w] != EDGE)) {
        out_pixels[w] = EDGE;
    }
    if ((in_pixels[e] >= t_low) && (out_pixels[e] != EDGE)) {
        out_pixels[e] = EDGE;
    }
    if ((in_pixels[sw] >= t_low) && (out_pixels[sw] != EDGE)) {
        out_pixels[sw] = EDGE;
    }
    if ((in_pixels[s] >= t_low) && (out_pixels[s] != EDGE)) {
        out_pixels[s] = EDGE;
    }
    if ((in_pixels[se] >= t_low) && (out_pixels[se] != EDGE)) {
        out_pixels[se] = EDGE;
    }
}

__global__
void cu_apply_hysteresis(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                        pixel_channel_t t_high, pixel_channel_t t_low, 
                        unsigned img_height, unsigned img_width)
{

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix; // TODO fix indexing
    if (iy <= img_height && ix <= img_width) {

        // apply high threshold
        if ((in_pixels[idx] > t_high) && (out_pixels[idx] != EDGE)) {
            out_pixels[idx] = EDGE;
        }
        // apply low threshold to neighbors
        trace_immed_neighbors(out_pixels, in_pixels, idx, t_low, img_width);
    }
}

void cu_detect_edges(pixel_t *orig_pixels, int rows, int cols, double kernel[KERNEL_SIZE][KERNEL_SIZE]) 
{
    pixel_t *in_pixels, *out_pixels;
    double *blurKernel, *cudaBlurKernel;
    int input_pixel_length = rows * cols;

    blurKernel = (double*) std::malloc(KERNEL_SIZE*KERNEL_SIZE*sizeof(double));

    for (int i = 0; i < KERNEL_SIZE; ++i) {
        for (int j = 0; j < KERNEL_SIZE; ++j) {
            blurKernel[i*KERNEL_SIZE + j] = kernel[i][j];
        }
    }

    /* allocate device memory */
    hipMalloc((void**) &in_pixels, input_pixel_length*sizeof(pixel_t));
    hipMalloc((void**) &out_pixels, input_pixel_length*sizeof(pixel_t));
    hipMalloc((void**) &cudaBlurKernel, KERNEL_SIZE*KERNEL_SIZE*sizeof(double));

    /* copy original pixels to GPU device as in_pixels*/
    hipMemcpy(in_pixels, orig_pixels, input_pixel_length*sizeof(pixel_t), hipMemcpyHostToDevice);
    hipMemcpy(cudaBlurKernel, blurKernel, KERNEL_SIZE*KERNEL_SIZE*sizeof(double), hipMemcpyHostToDevice);

      
    cu_apply_gaussian_filter<<<(rows*cols)/1024, 1024>>>(in_pixels, out_pixels, rows, cols, cudaBlurKernel);
    //cu_compute_intensity_gradient();
    //cu_suppress_non_max();
    //cu_apply_double_threshold();
    //cu_apply_hysteresis(pixel_t *out_pixels, pixel_t *in_pixels, pixel_t hi_thld, pixel_t lo_thld);

    /* copy blurred pixels from GPU device back to host as out_pixels*/
    hipMemcpy(orig_pixels, out_pixels, input_pixel_length * sizeof(pixel_t), hipMemcpyDeviceToHost);

    std::free(blurKernel);
    hipFree(cudaBlurKernel);
    hipFree(in_pixels);
    hipFree(out_pixels);
}

