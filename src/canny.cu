#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include "ed_pixel.h"

#define EDGE 0xFFFF

__device__ void trace_immed_neighbors(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                                     unsigned idx, pixel_channel_t t_low, unsigned img_width);

__global__
void cu_apply_hysteresis(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                        pixel_channel_t t_high, pixel_channel_t t_low, 
                        unsigned img_height, unsigned img_width)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix; // TODO fix indexing
    if (iy <= img_height && ix <= img_width) {

        // apply high threshold
        if ((in_pixels[idx] > t_high) && (out_pixels[idx] != EDGE)) {
            out_pixels[idx] = EDGE;
        }
        // apply low threshold to neighbors
        trace_immed_neighbors(out_pixels, in_pixels, idx, t_low, img_width);
    }
}

__device__
void trace_immed_neighbors(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                            unsigned idx, pixel_channel_t t_low, unsigned img_width)
{
    /* directions representing indices of neighbors */
    unsigned n, s, e, w;
    unsigned nw, ne, sw, se;

    /* get indices */
    n = idx - img_width;
    nw = n - 1;
    ne = n + 1;
    s = idx + img_width;
    sw = s - 1;
    se = s + 1;
    w = idx - 1;
    e = idx + 1;

    if ((in_pixels[nw] >= t_low) && (out_pixels[nw] != EDGE)) {
        out_pixels[nw] = EDGE;
    }
    if ((in_pixels[n] >= t_low) && (out_pixels[n] != EDGE)) {
        out_pixels[n] = EDGE;
    }
    if ((in_pixels[ne] >= t_low) && (out_pixels[ne] != EDGE)) {
        out_pixels[ne] = EDGE;
    }
    if ((in_pixels[w] >= t_low) && (out_pixels[w] != EDGE)) {
        out_pixels[w] = EDGE;
    }
    if ((in_pixels[e] >= t_low) && (out_pixels[e] != EDGE)) {
        out_pixels[e] = EDGE;
    }
    if ((in_pixels[sw] >= t_low) && (out_pixels[sw] != EDGE)) {
        out_pixels[sw] = EDGE;
    }
    if ((in_pixels[s] >= t_low) && (out_pixels[s] != EDGE)) {
        out_pixels[s] = EDGE;
    }
    if ((in_pixels[se] >= t_low) && (out_pixels[se] != EDGE)) {
        out_pixels[se] = EDGE;
    }
}

