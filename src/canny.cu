#include "hip/hip_runtime.h"

#include "canny.h"
#define _USE_MATH_DEFINES
#include <math.h>
#define EDGE 0xFFFF

__global__
void cu_apply_gaussian_filter(pixel_t *in_pixels, pixel_t *out_pixels, int rows, int cols, double kernel[KERNEL_SIZE][KERNEL_SIZE]) 
{
    int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixNum >= 0 && pixNum < rows * cols) {
   
        double kernelSum;
        double redPixelVal;
        double greenPixelVal;
        double bluePixelVal;

        //Apply Kernel to image
        for (int i = 0; i < KERNEL_SIZE; ++i) {
            for (int j = 0; j < KERNEL_SIZE; ++j) {                   
                //check edge cases, if within bounds, apply filter
                if (((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) >= 0)
                    && ((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) <= rows*cols-1)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) >= 0)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) <= (cols-1))) {

                    redPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].red;
                    greenPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].green;
                    bluePixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].blue;
                    kernelSum += kernel[i][j];
                }
            }
        }
        out_pixels[pixNum].red = redPixelVal / kernelSum;
        out_pixels[pixNum].green = greenPixelVal / kernelSum;
        out_pixels[pixNum].blue = bluePixelVal / kernelSum;
        redPixelVal = 0;
        greenPixelVal = 0;
        bluePixelVal = 0;
        kernelSum = 0;
     
    }
}

__device__
void trace_immed_neighbors(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                            unsigned idx, pixel_channel_t t_low, unsigned img_width)
{
    /* directions representing indices of neighbors */
    unsigned n, s, e, w;
    unsigned nw, ne, sw, se;

    /* get indices */
    n = idx - img_width;
    nw = n - 1;
    ne = n + 1;
    s = idx + img_width;
    sw = s - 1;
    se = s + 1;
    w = idx - 1;
    e = idx + 1;

    if ((in_pixels[nw] >= t_low) && (out_pixels[nw] != EDGE)) {
        out_pixels[nw] = EDGE;
    }
    if ((in_pixels[n] >= t_low) && (out_pixels[n] != EDGE)) {
        out_pixels[n] = EDGE;
    }
    if ((in_pixels[ne] >= t_low) && (out_pixels[ne] != EDGE)) {
        out_pixels[ne] = EDGE;
    }
    if ((in_pixels[w] >= t_low) && (out_pixels[w] != EDGE)) {
        out_pixels[w] = EDGE;
    }
    if ((in_pixels[e] >= t_low) && (out_pixels[e] != EDGE)) {
        out_pixels[e] = EDGE;
    }
    if ((in_pixels[sw] >= t_low) && (out_pixels[sw] != EDGE)) {
        out_pixels[sw] = EDGE;
    }
    if ((in_pixels[s] >= t_low) && (out_pixels[s] != EDGE)) {
        out_pixels[s] = EDGE;
    }
    if ((in_pixels[se] >= t_low) && (out_pixels[se] != EDGE)) {
        out_pixels[se] = EDGE;
    }
}

__global__
void cu_apply_hysteresis(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                        pixel_channel_t t_high, pixel_channel_t t_low, 
                        unsigned img_height, unsigned img_width)
{

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix; // TODO fix indexing
    if (iy <= img_height && ix <= img_width) {

        // apply high threshold
        if ((in_pixels[idx] > t_high) && (out_pixels[idx] != EDGE)) {
            out_pixels[idx] = EDGE;
        }
        // apply low threshold to neighbors
        trace_immed_neighbors(out_pixels, in_pixels, idx, t_low, img_width);
    }
}

void cu_detect_edges(pixel_t *orig_pixels, int rows, int cols, double kernel[KERNEL_SIZE][KERNEL_SIZE]) 
{
    pixel_t *in_pixels, *out_pixels;
    int input_pixel_length = rows * cols;

    /* allocate device memory */
    hipMalloc((void**) &in_pixels, input_pixel_length*sizeof(pixel_t));
    hipMalloc((void**) &out_pixels, input_pixel_length*sizeof(pixel_t));

    /* copy original pixels to GPU device as in_pixels*/
    hipMemcpy(in_pixels, orig_pixels, input_pixel_length*sizeof(pixel_t), hipMemcpyHostToDevice);
    hipMemcpy(out_pixels, orig_pixels, input_pixel_length*sizeof(pixel_t), hipMemcpyHostToDevice);
      
    cu_apply_gaussian_filter<<<(rows*cols)/1024, 1024>>>(in_pixels, out_pixels, rows, cols, kernel);
    //cu_compute_intensity_gradient();
    //cu_suppress_non_max();
    //cu_apply_double_threshold();
    //cu_apply_hysteresis(pixel_t *out_pixels, pixel_t *in_pixels, pixel_t hi_thld, pixel_t lo_thld);

    /* copy blurred pixels from GPU device back to host as out_pixels*/
    hipMemcpy(orig_pixels, out_pixels, input_pixel_length * sizeof(pixel_t), hipMemcpyDeviceToHost);

    hipFree(in_pixels);
    hipFree(out_pixels);
}

