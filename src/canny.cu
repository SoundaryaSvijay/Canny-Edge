#include "hip/hip_runtime.h"

#include <vector>
#include <math.h>
#include "canny.h"

#define _USE_MATH_DEFINES
#define STRONG_EDGE 0xFFFF
#define NON_EDGE 0x0

// TODO handle overlapping pixels (strong edges, neighbors)
// TODO nice viso c

__global__
void cu_apply_gaussian_filter(pixel_t *in_pixels, pixel_t *out_pixels, int rows, int cols, double *in_kernel) 
{
    //copy kernel array from global memory to a shared array
    __shared__ double kernel[KERNEL_SIZE][KERNEL_SIZE];
    for (int i = 0; i < KERNEL_SIZE; ++i) {
        for (int j = 0; j < KERNEL_SIZE; ++j) {
            kernel[i][j] = in_kernel[i*KERNEL_SIZE + j];
        }
    }
    
    __syncthreads();

    //determine id of thread which corresponds to an individual pixel
    int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixNum >= 0 && pixNum < rows * cols) {
   
        double kernelSum;
        double redPixelVal;
        double greenPixelVal;
        double bluePixelVal;

        //Apply Kernel to each pixel of image
        for (int i = 0; i < KERNEL_SIZE; ++i) {
            for (int j = 0; j < KERNEL_SIZE; ++j) {    
            
                //check edge cases, if within bounds, apply filter
                if (((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) >= 0)
                    && ((pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)) <= rows*cols-1)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) >= 0)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE-1)/2)) <= (cols-1))) {

                    redPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].red;
                    greenPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].green;
                    bluePixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2))*cols) + j - ((KERNEL_SIZE - 1) / 2)].blue;
                    kernelSum += kernel[i][j];
                }
            }
        }
        
        //update output image
        out_pixels[pixNum].red = redPixelVal / kernelSum;
        out_pixels[pixNum].green = greenPixelVal / kernelSum;
        out_pixels[pixNum].blue = bluePixelVal / kernelSum;
    }
}

//*****************************************************************************************
// CUDA Hysteresis Implementation
//*****************************************************************************************

///
/// \brief This is a helper function that runs on the GPU.
///
/// It checks if the eight immediate neighbors of a pixel at a given index are above
/// a low threshold, and if they are, sets them to strong edges. This effectively
/// connects the edges.
///
__device__
void trace_immed_neighbors(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                            unsigned idx, pixel_channel_t t_low, unsigned img_width)
{
    /* directions representing indices of neighbors */
    unsigned n, s, e, w;
    unsigned nw, ne, sw, se;

    /* get indices */
    n = idx - img_width;
    nw = n - 1;
    ne = n + 1;
    s = idx + img_width;
    sw = s - 1;
    se = s + 1;
    w = idx - 1;
    e = idx + 1;

    if (in_pixels[nw] >= t_low) {
        out_pixels[nw] = STRONG_EDGE;
    }
    if (in_pixels[n] >= t_low) {
        out_pixels[n] = STRONG_EDGE;
    }
    if (in_pixels[ne] >= t_low) {
        out_pixels[ne] = STRONG_EDGE;
    }
    if (in_pixels[w] >= t_low) {
        out_pixels[w] = STRONG_EDGE;
    }
    if (in_pixels[e] >= t_low) {
        out_pixels[e] = STRONG_EDGE;
    }
    if (in_pixels[sw] >= t_low) {
        out_pixels[sw] = STRONG_EDGE;
    }
    if (in_pixels[s] >= t_low) {
        out_pixels[s] = STRONG_EDGE;
    }
    if (in_pixels[se] >= t_low) {
        out_pixels[se] = STRONG_EDGE;
    }
}

///
/// \brief CUDA implementation of Canny hysteresis high thresholding.
///
/// This kernel is the first pass in the parallel hysteresis step.
/// It launches a thread for every pixel and checks if the value of that pixel
/// is above a high threshold. If it is, the thread marks it as a strong edge (set to 1)
/// in a pixel map and sets the value to the channel max. If it is not, the thread sets
/// the pixel map at the index to 0 and zeros the output buffer space at that index.
///
/// The output of this step is a mask of strong edges and an output buffer with white values
/// at the mask indices which are set.
///
__global__
void cu_hysteresis_high(unsigned *strong_edge_mask, pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, 
                        pixel_channel_t t_high, unsigned img_height, unsigned img_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (img_height * img_width)) {
        /* apply high threshold */
        if (in_pixels[idx] > t_high) {
            strong_edge_mask[idx] = 1;
            out_pixels[idx] = STRONG_EDGE;
        } else {
            strong_edge_mask[idx] = 0;
            out_pixels[idx] = NON_EDGE;
        }
    }
}

///
/// \brief CUDA implementation of Canny hysteresis low thresholding.
///
/// This kernel is the second pass in the parallel hysteresis step. 
/// It launches a thread for every pixel, but skips the first and last rows and columns.
/// For surviving threads, the pixel at the thread ID index is checked to see if it was 
/// previously marked as a strong edge in the first pass. If it was, the thread checks 
/// their eight immediate neighbors and connects them (marks them as strong edges)
/// if the neighbor is above the low threshold.
///
/// The output of this step is an output buffer with both "strong" and "connected" edges
/// set to whtie values. This is the final edge detected image.
///
__global__
void cu_hysteresis_low(pixel_channel_t *out_pixels, pixel_channel_t *in_pixels, unsigned *strong_edge_mask,
                        unsigned t_low, unsigned img_height, unsigned img_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ((idx > img_width)                               /* skip first row */
        && (idx < (img_height * img_width) - img_width) /* skip last row */
        && ((idx % img_width) < (img_width - 1))        /* skip last column */
        && ((idx % img_width) > (0)) )                  /* skip first column */
    {
        if (1 == strong_edge_mask[idx]) { /* if this pixel was previously found to be a strong edge */
            trace_immed_neighbors(out_pixels, in_pixels, idx, t_low, img_width);
        }
    }
}

void cu_test_hysteresis(pixel_channel_t *in, pixel_channel_t *out, unsigned rows, unsigned cols)
{
    pixel_channel_t *in_pixels, *out_pixels;
    unsigned *idx_map;

    /* allocate device memory */
    hipMalloc((void**) &in_pixels, rows*cols*sizeof(pixel_channel_t));
    hipMalloc((void**) &out_pixels, rows*cols*sizeof(pixel_channel_t));
    hipMalloc((void**) &idx_map, rows*cols*sizeof(idx_map[0]));

    /* copy original pixels to GPU device as in_pixels*/
    hipMemcpy(in_pixels, in, rows*cols*sizeof(pixel_channel_t), hipMemcpyHostToDevice);
      
    pixel_channel_t t_high = 0xFCC;
    pixel_channel_t t_low = 0x1FF;

    /* create task stream to sequence kernels */
    hipStream_t stream;
    hipStreamCreate(&stream);

    /* launch kernels */
    cu_hysteresis_high<<<(rows*cols)/1024, 1024, 0, stream>>>(idx_map, out_pixels, in_pixels, t_high, rows, cols);
    cu_hysteresis_low<<<(rows*cols)/1024, 1024, 0, stream>>>(out_pixels, in_pixels, idx_map, t_low, rows, cols);

    /* copy blurred pixels from GPU device back to host as out_pixels*/
    hipMemcpy(out, out_pixels, rows*cols*sizeof(pixel_channel_t), hipMemcpyDeviceToHost);

    hipFree(in_pixels);
    hipFree(out_pixels);
    hipFree(idx_map);
}

void cu_detect_edges(pixel_t *orig_pixels, int rows, int cols, double kernel[KERNEL_SIZE][KERNEL_SIZE]) 
{
    pixel_t *in_pixels, *out_pixels;
    double *blurKernel, *cudaBlurKernel;
    int input_pixel_length = rows * cols;

    blurKernel = (double*) std::malloc(KERNEL_SIZE*KERNEL_SIZE*sizeof(double));

    for (int i = 0; i < KERNEL_SIZE; ++i) {
        for (int j = 0; j < KERNEL_SIZE; ++j) {
            blurKernel[i*KERNEL_SIZE + j] = kernel[i][j];
        }
    }

    /* allocate device memory */
    hipMalloc((void**) &in_pixels, input_pixel_length*sizeof(pixel_t));
    hipMalloc((void**) &out_pixels, input_pixel_length*sizeof(pixel_t));
    hipMalloc((void**) &cudaBlurKernel, KERNEL_SIZE*KERNEL_SIZE*sizeof(double));

    /* copy original pixels to GPU device as in_pixels*/
    hipMemcpy(in_pixels, orig_pixels, input_pixel_length*sizeof(pixel_t), hipMemcpyHostToDevice);
    hipMemcpy(cudaBlurKernel, blurKernel, KERNEL_SIZE*KERNEL_SIZE*sizeof(double), hipMemcpyHostToDevice);

    cu_apply_gaussian_filter<<<(rows*cols)/1024, 1024>>>(in_pixels, out_pixels, rows, cols, cudaBlurKernel);
    //cu_compute_intensity_gradient();
    //cu_suppress_non_max();
    //cu_apply_double_threshold();
    //pixel_channel_t t_high = 0xFCC;
    //pixel_channel_t t_low = 0xF5;
    //cu_apply_hysteresis<<<(rows*cols)/1024, 1024>>>(out_pixels, in_pixels, t_high, t_low, rows, cols);

    /* copy blurred pixels from GPU device back to host as out_pixels*/
    hipMemcpy(orig_pixels, out_pixels, input_pixel_length * sizeof(pixel_t), hipMemcpyDeviceToHost);

    std::free(blurKernel);
    hipFree(cudaBlurKernel);
    hipFree(in_pixels);
    hipFree(out_pixels);
}

